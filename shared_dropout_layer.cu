#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/shared_dropout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SharedDropoutForward(const int n, const Dtype* in,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (mask[index] > threshold) * scale;
  }
}

template <typename Dtype>
void SharedDropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  if (this->phase_ == TRAIN) {
    unsigned int* mask =
        static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
    caffe_gpu_rng_uniform(count, mask);
    // set thresholds
    for(int i = 0; i < top.size(); ++i) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* top_data = top[i]->mutable_gpu_data();
      if (scale_train_) {
        // NOLINT_NEXT_LINE(whitespace/operators)
        SharedDropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count),
          CAFFE_CUDA_NUM_THREADS>>>(
            count, bottom_data, mask, uint_thres_, scale_, top_data);
      } else {
        // NOLINT_NEXT_LINE(whitespace/operators)
        SharedDropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count),
          CAFFE_CUDA_NUM_THREADS>>>(
            count, bottom_data, mask, uint_thres_, 1.f, top_data);
      }
      CUDA_POST_KERNEL_CHECK;
    }
  } else {
    for(int i = 0; i < top.size(); ++i) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* top_data = top[i]->mutable_gpu_data();
      caffe_copy(count, bottom_data, top_data);
      if (!scale_train_) {
        caffe_gpu_scal<Dtype>(count, 1. / scale_, top_data);
      }
    }
  }
}

template <typename Dtype>
__global__ void SharedDropoutBackward(const int n, const Dtype* in_diff,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * (mask[index] > threshold);
  }
}

template <typename Dtype>
void SharedDropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    if (this->phase_ == TRAIN) {
      const unsigned int* mask =
          static_cast<const unsigned int*>(rand_vec_.gpu_data());
      const int count = bottom[0]->count();
      for(int i = 0; i < top.size(); ++i) {
        const Dtype* top_diff = top[i]->gpu_diff();
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        if (scale_train_) {
          // NOLINT_NEXT_LINE(whitespace/operators)
          SharedDropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
            CAFFE_CUDA_NUM_THREADS>>>(
              count, top_diff, mask, uint_thres_, scale_, bottom_diff);
        } else {
          // NOLINT_NEXT_LINE(whitespace/operators)
          SharedDropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
            CAFFE_CUDA_NUM_THREADS>>>(
             count, top_diff, mask, uint_thres_, 1.f, bottom_diff);
        }
        CUDA_POST_KERNEL_CHECK;
      }
    } else {
      for(int i = 0; i < top.size(); ++i) {
        const Dtype* top_diff = top[i]->gpu_diff();
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        caffe_copy(top[0]->count(), top_diff, bottom_diff);
        if (!scale_train_) {
          caffe_gpu_scal<Dtype>(top[0]->count(), 1. / scale_, bottom_diff);
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SharedDropoutLayer);

}  // namespace caffe
